#include "hip/hip_runtime.h"
#include "encabezados.h"

// ===================================================================================
// HOST
// ===================================================================================

void potential_wca_hst(float sigma, float dist_inv, float &potential, float &normal_force){
    float sor = dist_inv*sigma;
    float sor6, sor12;
    sor6 = sor*sor*sor;
    sor6 = sor6*sor6;
    sor12=sor6*sor6;

    potential += EPS * 4 * (sor12 - sor6) + 1;
    normal_force = - EPS * 4 * (12 * sor12 - 6 * sor6) * dist_inv;

}

void get_forces_same_hst(char type, float3 *rr_vec, float3 *ff_vec, float *vir_vec, 
         float *pot_vec, int *nocup_vec, int *cell_vec, parametros pars)
{
  static float3 rrm, rrn, drr, ff_pair, ffm;
  static float cutoff2, side, side_inv, cutoff, virial, potential, sigma,
                cell_side_inv, dist_inv, dist2, sor, sor6, normal_force;
  static int ii, jj, kk, i_del, j_del, k_del, iip, jjp, kkp, tag, ncell,
	     cell_index, mm, nn, tag_init, tag_end, ntags, nocup, ngrain, nrange;

  if (type == 'b')
  { 
    sigma = pars.sigma_big;
    ngrain = pars.ngrain_big;
    cell_side_inv = 1.0f/pars.cell_side_big;
    ntags = pars.ntags_big;
    ncell = pars.ncell_big;
    nrange = pars.nrange_bb;
  }
  else
  { 
    sigma = pars.sigma_sml;
    ngrain = pars.ngrain_sml;
    cell_side_inv = 1.0f/pars.cell_side_sml;
    ntags = pars.ntags_sml;
    ncell = pars.ncell_sml;
    nrange = pars.nrange_ss;
  }

    //wca
    cutoff = 1.122462048f * sigma;

    cutoff2 = cutoff*cutoff;
    side = pars.side;
    side_inv = 1.0f/side;


  for (mm = 0; mm < ngrain; mm++)
  {
    // fetch

    rrm = rr_vec[mm];
    ffm = ff_vec[mm];
    virial = vir_vec[mm];
    potential = pot_vec[mm];

    // calculate position of mm in the cells

    ii = (int) (cell_side_inv*rrm.x);
    jj = (int) (cell_side_inv*rrm.y);
    kk = (int) (cell_side_inv*rrm.z);
    if (ii == ncell) ii--;
    if (jj == ncell) jj--;
    if (kk == ncell) kk--;

    // run through neighbor cells

    for (i_del = -nrange; i_del <= nrange; i_del++) 
        for (j_del = -nrange; j_del <= nrange; j_del++)
            for (k_del = -nrange; k_del <= nrange; k_del++)
    {
      iip = (ii + i_del + ncell)%ncell;
      jjp = (jj + j_del + ncell)%ncell;
      kkp = (kk + k_del + ncell)%ncell;
      cell_index = iip + ncell*(jjp + ncell*kkp);
      nocup = nocup_vec[cell_index];
      tag_init = cell_index*ntags;
      tag_end = tag_init + nocup;
    
      // check grain in cell

      for (tag = tag_init; tag < tag_end; tag++) 
      {
        nn = cell_vec[tag];
        if (nn == mm) continue;
         
        // fetch another

        rrn = rr_vec[nn];

        // distance
  
        drr.x = rrn.x - rrm.x;
        drr.y = rrn.y - rrm.y;  
        drr.z = rrn.z - rrm.z;  

        // periodic boundary conditions

        drr.x -= side*floor(side_inv*drr.x + 0.5f);
        drr.y -= side*floor(side_inv*drr.y + 0.5f);
        drr.z -= side*floor(side_inv*drr.z + 0.5f);
  
        // distance and normal force
        
        dist2 = drr.x*drr.x + drr.y*drr.y + drr.z*drr.z;
        if (dist2 < cutoff2)
        {
            dist_inv = sqrt(1.0f/dist2);
            potential_wca_hst(sigma, dist_inv, potential, normal_force);
              
          // calculate normal force
              
          ff_pair.x = normal_force*dist_inv*drr.x;
          ff_pair.y = normal_force*dist_inv*drr.y;
          ff_pair.z = normal_force*dist_inv*drr.z;

          // sum forces and virial

          ffm.x += ff_pair.x;
          ffm.y += ff_pair.y;
          ffm.z += ff_pair.z;

          virial -= 0.5f*(drr.x*ff_pair.x + drr.y*ff_pair.y + drr.z*ff_pair.z);
        }
      }
    }
    
    // save 

    ff_vec[mm] = ffm;
    vir_vec[mm] = virial;
    pot_vec[mm] = potential;
  }
}

// ===================================================================================

void get_forces_diff_hst(char type, char type_other, float3 *rr_vec, float3 *ff_vec, 
         float3 *rr_other_vec, float *vir_vec, float *pot_vec, int *nocup_other_vec, 
         int *cell_other_vec, parametros pars)
{
  static float3 rrm, rrn, drr, ff_pair, ffm;
  static float cutoff2, side, side_inv, cutoff, sigma, virial, potential,
	       cell_side_inv, dist_inv, dist2, sor, sor6, normal_force;
  static int ii, jj, kk, i_del, j_del, k_del, iip, jjp, kkp, tag, ncell, mm, nn, 
             cell_index, tag_init, tag_end, ntags, nocup, ngrain, nrange;

  side = pars.side;
  side_inv = 1.0f/side;

  if (type == 'b')
  {
    ngrain = pars.ngrain_big;
    cell_side_inv = 1.0f/pars.cell_side_sml;
    sigma = 0.5f*(pars.sigma_big + pars.sigma_sml);
    ntags = pars.ntags_sml;
    ncell = pars.ncell_sml;
    nrange = pars.nrange_bs;
  }
  else
  {
    ngrain = pars.ngrain_sml;
    cell_side_inv = 1.0f/pars.cell_side_big;
    sigma = 0.5f*(pars.sigma_sml + pars.sigma_big);
    ntags = pars.ntags_big;
    ncell = pars.ncell_big;
    nrange = pars.nrange_sb;
  }

    //wca
    cutoff = 1.122462048f * sigma;

    cutoff2 = cutoff*cutoff;
    side = pars.side;
    side_inv = 1.0f/side;


  for (mm = 0; mm < ngrain; mm++)
  {
    // fetch

    rrm = rr_vec[mm];
    ffm = ff_vec[mm];
    virial = vir_vec[mm];
    potential = pot_vec[mm];

    // calculate cell for mm

    ii = (int) (cell_side_inv*rrm.x);
    jj = (int) (cell_side_inv*rrm.y);
    kk = (int) (cell_side_inv*rrm.z);
    if (ii == ncell) ii--;
    if (jj == ncell) jj--;
    if (kk == ncell) kk--;

    // run through neighbor cells

    for (i_del = -nrange; i_del <= nrange; i_del++) 
        for (j_del = -nrange; j_del <= nrange; j_del++)
            for (k_del = -nrange; k_del <= nrange; k_del++)
    {
      iip = (ii + i_del + ncell)%ncell;
      jjp = (jj + j_del + ncell)%ncell;
      kkp = (kk + k_del + ncell)%ncell;
      cell_index = iip + ncell*(jjp + ncell*kkp);
      nocup = nocup_other_vec[cell_index];
      tag_init = cell_index*ntags;
      tag_end = tag_init + nocup;
    
      // check others

      for (tag = tag_init; tag < tag_end; tag++) 
      {
        nn = cell_other_vec[tag];

        // fetch again

        rrn = rr_other_vec[nn];
        
        // coordinate differences
  
        drr.x = rrn.x - rrm.x;
        drr.y = rrn.y - rrm.y;  
        drr.z = rrn.z - rrm.z;  

        // periodic boundary conditions

        drr.x -= side*floor(side_inv*drr.x + 0.5f);	// per. bound. cond.
        drr.y -= side*floor(side_inv*drr.y + 0.5f);
        drr.z -= side*floor(side_inv*drr.z + 0.5f);
    
        // distance and normal force

        dist2 = drr.x*drr.x + drr.y*drr.y + drr.z*drr.z;
        if (dist2 < cutoff2)
        {
            dist_inv = sqrt(1.0f/dist2);
            potential_wca_hst(sigma, dist_inv, potential, normal_force);

          // calculate normal force
              
          ff_pair.x = normal_force*dist_inv*drr.x;
          ff_pair.y = normal_force*dist_inv*drr.y;
          ff_pair.z = normal_force*dist_inv*drr.z;

          // sum forces

          ffm.x += ff_pair.x;
          ffm.y += ff_pair.y;
          ffm.z += ff_pair.z;

          virial -= 0.5f*(drr.x*ff_pair.x + drr.y*ff_pair.y + drr.z*ff_pair.z);
        }
      }
    }

    // save

    ff_vec[mm] = ffm; 
    vir_vec[mm] = virial; 
    pot_vec[mm] = potential; 
  } 
  return;
}

// ===================================================================================
// DEVICE
// ===================================================================================

__device__ void potential_wca_dev(float sigma, float dist_inv, float &potential, float &normal_force) {
    float sor = dist_inv * sigma;
    float sor6, sor12;
    sor6 = sor * sor * sor;
    sor6 = sor6 * sor6;
    sor12 = sor6 * sor6;

    potential += EPS * 4 * (sor12 - sor6) + 1;
    normal_force = -EPS * 4 * (12 * sor12 - 6 * sor6) * dist_inv;
}

__global__ void get_forces_same_dev(char type, float3 *rr_vec, float3 *ff_vec, 
                    float *vir_vec, float *pot_vec, int *nocup_vec, 
                    int *cell_vec, parametros pars)
{
  float3 rrm, rrn, drr, ffm, ff_pair;
  float cutoff2, side, side_inv, cutoff, virial, potential, sigma, cell_side_inv,
         dist_inv, dist2, normal_force, temp;
  int ii, jj, kk, i_del, j_del, k_del, iip, jjp, kkp, tag, ncell, cell_index, mm, nn,
      tag_init, tag_end, ntags, nocup, ngrain, nrange;

  if (type == 'b')
  { 
    sigma = pars.sigma_big;
    ngrain = pars.ngrain_big;
    cell_side_inv = 1.0f/pars.cell_side_big;
    ntags = pars.ntags_big;
    ncell = pars.ncell_big;
    nrange = pars.nrange_bb;
    temp = pars.temp_set;
  }
  else
  { 
    sigma = pars.sigma_sml;
    ngrain = pars.ngrain_sml;
    cell_side_inv = 1.0f/pars.cell_side_sml;
    ntags = pars.ntags_sml;
    ncell = pars.ncell_sml;
    nrange = pars.nrange_ss;
    temp = pars.temp_set;
  }

    //wca
  cutoff = 1.122462048f * sigma;

  cutoff2 = cutoff*cutoff;
  side = pars.side;
  side_inv = 1.0f/side;


  mm = threadIdx.x + blockIdx.x*blockDim.x;

  if (mm < ngrain)
  {
    // fetch

    rrm = rr_vec[mm];
    ffm = ff_vec[mm];
    virial = vir_vec[mm];
    potential = pot_vec[mm];

    // calculate cell for mm

    ii = (int) (cell_side_inv*rrm.x);
    jj = (int) (cell_side_inv*rrm.y);
    kk = (int) (cell_side_inv*rrm.z);
    if (ii == ncell) ii--;
    if (jj == ncell) jj--;
    if (kk == ncell) kk--;

    // run through neighbor cells

    for (i_del = -nrange; i_del <= nrange; i_del++) 
        for (j_del = -nrange; j_del <= nrange; j_del++)
            for (k_del = -nrange; k_del <= nrange; k_del++)
    {
      iip = (ii + i_del + ncell)%ncell;
      jjp = (jj + j_del + ncell)%ncell;
      kkp = (kk + k_del + ncell)%ncell;
      cell_index = iip + ncell*(jjp + ncell*kkp);
      nocup = nocup_vec[cell_index];
      tag_init = cell_index*ntags;
      tag_end = tag_init + nocup;
    
      // check another grain

      for (tag = tag_init; tag < tag_end; tag++) 
      {
        nn = cell_vec[tag];
        if (nn == mm) continue;
         
        // another fetch

        rrn = rr_vec[nn];

        // coordinate differences
 
        drr.x = rrn.x - rrm.x;
        drr.y = rrn.y - rrm.y;  
        drr.z = rrn.z - rrm.z;  

        // periodic boundary conditions

        drr.x -= side*floor(side_inv*drr.x + 0.5f);	// per. bound. cond.
        drr.y -= side*floor(side_inv*drr.y + 0.5f);
        drr.z -= side*floor(side_inv*drr.z + 0.5f);
  
        // distance

        dist2 = drr.x*drr.x + drr.y*drr.y + drr.z*drr.z;
        if (dist2 < cutoff2)
        {
          dist_inv = sqrt(1.0f/dist2);
          potential_wca_dev(sigma, dist_inv, potential, normal_force);
              
          // calculate normal force
              
          ff_pair.x = normal_force*dist_inv*drr.x;
          ff_pair.y = normal_force*dist_inv*drr.y;
          ff_pair.z = normal_force*dist_inv*drr.z;

          // sum forces and virial

          ffm.x += ff_pair.x;
          ffm.y += ff_pair.y;
          ffm.z += ff_pair.z;

          virial -= 0.5f*(drr.x*ff_pair.x + drr.y*ff_pair.y + drr.z*ff_pair.z);
        }
      }
    }

    // save

    ff_vec[mm] = ffm;
    vir_vec[mm] = virial;
    pot_vec[mm] = potential;
  }
}

// ===================================================================================

__global__ void get_forces_diff_dev(char type, char type_other, float3 *rr_vec, 
                    float3 *ff_vec, float3 *rr_other_vec, float *vir_vec, 
                    float *pot_vec, int *nocup_other_vec, int *cell_other_vec,
                    parametros pars)
{
  float3 rrm, rrn, drr, ff_pair, ffm;
  float cutoff2, side, side_inv, cutoff, sigma, virial, potential, cell_side_inv,
	 dist_inv, dist2, normal_force, temp;
  int ii, jj, kk, i_del, j_del, k_del, iip, jjp, kkp, tag, ncell, mm, nn, cell_index,
      tag_init, tag_end, ntags, nocup, ngrain, nrange;

  if (type == 'b')
  {
    ngrain = pars.ngrain_big;
    cell_side_inv = 1.0f/pars.cell_side_sml;
    sigma = 0.5f*(pars.sigma_big + pars.sigma_sml);
    ntags = pars.ntags_sml;
    ncell = pars.ncell_sml;
    nrange = pars.nrange_bs;
    temp = pars.temp_set;

  }
  else
  {
    ngrain = pars.ngrain_sml;
    cell_side_inv = 1.0f/pars.cell_side_big;
    sigma = 0.5f*(pars.sigma_sml + pars.sigma_big);
    ntags = pars.ntags_big;
    ncell = pars.ncell_big;
    nrange = pars.nrange_sb;
    temp = pars.temp_set;

  }

  //wca
  cutoff = 1.122462048f * sigma;

  cutoff2 = cutoff*cutoff;
  side = pars.side;
  side_inv = 1.0f/side;

  // calcula interacciones sobre grandes

  mm = threadIdx.x + blockIdx.x*blockDim.x;

  if (mm < ngrain)
  {
    // fetch

    rrm = rr_vec[mm];
    ffm = ff_vec[mm];
    virial = vir_vec[mm];
    potential = pot_vec[mm];

    // calcula posicion de mm

    ii = (int) (cell_side_inv*rrm.x);
    jj = (int) (cell_side_inv*rrm.y);
    kk = (int) (cell_side_inv*rrm.z);
    if (ii == ncell) ii--;
    if (jj == ncell) jj--;
    if (kk == ncell) kk--;

    // recorre vecindario

    for (i_del = -nrange; i_del <= nrange; i_del++) 
        for (j_del = -nrange; j_del <= nrange; j_del++)
            for (k_del = -nrange; k_del <= nrange; k_del++)
    {
      iip = (ii + i_del + ncell)%ncell;
      jjp = (jj + j_del + ncell)%ncell;
      kkp = (kk + k_del + ncell)%ncell;
      cell_index = iip + ncell*(jjp + ncell*kkp);
      nocup = nocup_other_vec[cell_index];
      tag_init = cell_index*ntags;
      tag_end = tag_init + nocup;
    
      // checa con granos de taman~o diferente

      for (tag = tag_init; tag < tag_end; tag++) 
      {
        nn = cell_other_vec[tag];
        
        // another fetch

        rrn = rr_other_vec[nn];
        
        // corrdinate differences
  
        drr.x = rrn.x - rrm.x;
        drr.y = rrn.y - rrm.y;  
        drr.z = rrn.z - rrm.z;  

        // periodic boundary conditions

        drr.x -= side*floor(side_inv*drr.x + 0.5f);	// per. bound. cond.
        drr.y -= side*floor(side_inv*drr.y + 0.5f);
        drr.z -= side*floor(side_inv*drr.z + 0.5f);
    
        // distance

        dist2 = drr.x*drr.x + drr.y*drr.y + drr.z*drr.z ;
        if (dist2 < cutoff2)
        {
          dist_inv = sqrt(1.0f/dist2);
          potential_wca_dev(sigma, dist_inv, potential, normal_force);
              
          // calcula las fuerzas normales para este par
              
          ff_pair.x = normal_force*dist_inv*drr.x;
          ff_pair.y = normal_force*dist_inv*drr.y;
          ff_pair.z = normal_force*dist_inv*drr.z;

          // suma a fuerzas, al potencial y al virial

          ffm.x += ff_pair.x;
          ffm.y += ff_pair.y;
          ffm.z += ff_pair.z;

          virial -= 0.5f*(drr.x*ff_pair.x + drr.y*ff_pair.y + drr.z*ff_pair.z);
        }
      }
    }

    // save

    ff_vec[mm] = ffm;
    vir_vec[mm] = virial;
    pot_vec[mm] = potential;
  }
}
